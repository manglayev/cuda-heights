#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
#include <array>

#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/sequence.h>
#include <thrust/universal_vector.h>
#include <thrust/execution_policy.h>

using namespace std;

/*
    Example of thrust zip reduce iterator
*/

void max_change(const thrust::universal_vector<int> &a, const thrust::universal_vector<int> &b)
{
    auto zip = thrust::make_zip_iterator(a.begin(), b.begin());

    auto transformation = [] __host__ __device__ (thrust::tuple<int, int> t)
    {
        return abs(thrust::get<0>(t) - thrust::get<1>(t));
    };

    auto transform = make_transform_iterator(zip, transformation);

    float m = thrust::reduce(thrust::device, transform, transform + a.size(), 0.0f, thrust::maximum<float>{});

    printf("max change % 2.f\n", m);

}

int main()
{
    // allocate vectors containing 2^28 elements
    thrust::universal_vector<int> a(1 << 8);
    thrust::universal_vector<int> b(1 << 8);

    thrust::sequence(a.begin(), a.end());
    thrust::sequence(b.rbegin(), b.rend());

    max_change(a, b);
}