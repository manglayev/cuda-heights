
#include <hip/hip_runtime.h>
#include <cstdio>
#include <iostream>
#include <array>

using namespace std;

/*
    Using a pointer to access data in array
*/

int main()
{
    cout << "main start;\n";
    constexpr size_t size = 5;
    std::array<int, size> a{1, 3, 5, 7, 9};

    int *a_p = a.data();

    printf("a[%d] = %d\n", 3, a[2]);
    printf("a_p[%d] = %d\n", 3, a_p[2]);
    cout << "main end;\n";
}