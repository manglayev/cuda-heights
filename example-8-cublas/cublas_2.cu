#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hipblas.h>
#include <hiprand.h>
#include "cublas_2_header.cuh"

//Verify the result 3
void verify_result(float *a, float *b, float *c)
{
  float temp;
  float epsilon = 0.001;;
  for(int i = 0; i < N; i++)
  {
    for(int j = 0; j < N; j++)
    {
      temp = 0;
      for(int k = 0; k < N; k++)
      {
        temp += a[k * N + i] * b[j * N + k];
      }
      assert(fabs(c[j * N + i] - temp) < epsilon);
    }    
  }
}
/*
//Print N x N matrix
*/
void printArray(float *a)
{
  // loop through array's rows
  for(int i = 0; i < N; i++)
  {
    if(a[i] >= 0)
      std::cout << ' ' << a[i] << ' ';
    else
      std::cout << a[i] << ' ';
  }
  std::cout << '\n'; // start new line of output
}

void wrapper()
{
  printf("STAGE 3 WRAPPER START\n");
  // Vector size
  size_t bytes = N * N * sizeof(float);
  //Declare vector pointers
  float *h_a, *h_b, *h_c;
  float *d_a, *d_b, *d_c;

  // Allocate memory
  h_a = (float*)malloc(bytes);
  h_b = (float*)malloc(bytes);
  h_c = (float*)malloc(bytes);
  
  hipMalloc(&d_a, bytes);
  hipMalloc(&d_b, bytes);
  hipMalloc(&d_c, bytes);

  //pseudorandom number generator
  hiprandGenerator_t prng;
  hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);
  
  //set the seed
  hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long)clock());

  //Fill the matrix with the random number on the device
  hiprandGenerateUniform(prng, d_a, N*N);
  hiprandGenerateUniform(prng, d_b, N*N);
  
  // Create and initialize a new context
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  float alpha = 1.0f;
  float beta = 0.0f;

  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_a, N, d_b, N, &beta, d_c, N);

  hipMemcpy(h_a, d_a, bytes, hipMemcpyDeviceToHost);
  hipMemcpy(h_b, d_b, bytes, hipMemcpyDeviceToHost);
  hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

  
  verify_result(h_a, h_b, h_c);
  //Print out the result
  printArray(h_c);

  //Clean up the created handle
  hipblasDestroy(handle);
  printf("STAGE 3 WRAPPER END\n");
}