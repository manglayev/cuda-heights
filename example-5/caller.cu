#include <iostream>

#include "hip/hip_runtime.h"


using namespace std;
__global__ void doubleArray(int *a, int N)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for(int i = index; i < N; i += stride)
    {
        a[i] = a[i] * 2;
    }
}

void print(int *a, int N)
{
    int c = 0;
    for(int i = 0; i < N; i++)
    {
        c++;
        cout<<a[i]<<" ";
        if (c % 8 == 0) cout <<"\n";
    }
}

int main()
{
    int N = 2 << 7;
    size_t size = N * sizeof(int);
    
    int h_a[N];    
    for(int i = 0; i < N; i++)
    {
        h_a[i] = 1;
    }

    int *d_a;
    hipMalloc((void**)&d_a, size);
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    
    doubleArray<<<2,4>>>(d_a, N);
    
    hipMemcpy(h_a, d_a, size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    
    hipFree(d_a);
    
    print(h_a, N);
}