#include "hip/hip_runtime.h"
#include "example-1-header.h"
#include ""
#include "hip/hip_runtime.h"

__global__ void kernel()
{
    printf("GPU kernel\n");
}

void caller()
{
    cout << "caller start;\n";
    kernel<<<1,1>>>();
    hipDeviceSynchronize();
    cout << "caller end;\n";
}