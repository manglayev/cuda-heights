#include "hip/hip_runtime.h"
#include <iostream>
#include ""
#include "hip/hip_runtime.h"

using namespace std;

__global__ void init(int *a, int N)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < N; i += stride)
    {
        a[i] = 1;
    }
}

__global__ void summ(int *a, int *b, int *c, int N)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for(int i = index; i < N; i+= stride)
    {
        c[i] = a[i] + b[i];
    }
}

void print(int *a, int N)
{
    int c = 0;
    for (int i = 0; i < N; i++)
    {
        cout << a[i] << " ";
        c++;
        if(c % 8 == 0) cout << "\n";
    }    
}

int main()
{
    int *a;
    int *b;
    int *c;

    const int N = 2 << 7;
    size_t size = N * sizeof(int);
    
    hipMallocManaged(&a, size);
    hipMallocManaged(&b, size);
    hipMallocManaged(&c, size);
    
    int deviceId;
    int numberOfSMs;

    hipGetDevice(&deviceId);
    hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

    hipMemPrefetchAsync(a, size, deviceId);
    hipMemPrefetchAsync(b, size, deviceId);
    hipMemPrefetchAsync(c, size, deviceId);

    hipStream_t stream_1;
    hipStream_t stream_2;
    hipStream_t stream_3;
    
    hipStreamCreate(&stream_1);
    hipStreamCreate(&stream_2);
    hipStreamCreate(&stream_3);    

    init<<<8, 32, 0, stream_1>>>(a, N);
    init<<<8, 32, 0, stream_2>>>(b, N);
    init<<<8, 32, 0, stream_3>>>(c, N);

    hipStreamDestroy(stream_1);
    hipStreamDestroy(stream_2);
    hipStreamDestroy(stream_3);

    summ<<<8, 32>>>(a, b, c, N);

    hipMemPrefetchAsync(c, size, hipCpuDeviceId);
    print(c, N);

    hipFree(a);
    hipFree(b);
    hipFree(c);
}