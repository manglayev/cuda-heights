
#include <hip/hip_runtime.h>
#include <cstdio>
#include <iostream>
#include <array>

using namespace std;

/*
    Example of iterator
*/

struct transform_iterator
{
    int *a;
    int operator[](int i)
    {
        return i * i;
    }
};

int main()
{
    cout << "main start;\n";
    std::array<int, 5> a{1, 3, 5, 7, 9};
    int *a_p = a.data();
    transform_iterator ti{a.data()};
    
    printf("ti[%d] = %d\n", 0, ti[0]);
    printf("ti[%d] = %d\n", 3, ti[3]);

    printf("a[%d] = %d\n", 0, a[0]);
    printf("a[%d] = %d\n", 3, a[3]);

    printf("a_p[%d] = %d\n", 0, a[0]);
    printf("a_p[%d] = %d\n", 3, a[3]);

    cout << "main end;\n";
}