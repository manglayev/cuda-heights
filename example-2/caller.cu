#include "hip/hip_runtime.h"
#include "example-2-header.h"
#include ""
#include "hip/hip_runtime.h"

__global__ void kernel(int *a, int *b, int *c, int N)
{
    //printf("GPU kernel start");
    int index = threadIdx.x;
    if (index < N)
    {
        c[index] = a[index] + b[index];
    }
    //printf("GPU kernel end");
}

void init(int *a, int N)
{
    for(int i = 0; i < N; i++)
    {
        a[i] = 1;
    }
}

void print(int *a, int N)
{
    for(int i = 0; i < N; i++)
    {
        cout<<a[i]<<" ";
    }
    cout << "\n";
}
void caller()
{
    cout<<"caller start\n";
    
    int *a;
    int *b;
    int *c;
    const int N = 1<<4;
    cout<<N<<"\n";
    size_t size = N * sizeof(int);

    hipMallocManaged(&a, size);
    hipMallocManaged(&b, size);
    hipMallocManaged(&c, size);

    init(a, N);
    init(b, N);

    kernel<<<1, N>>>(a, b, c, N);

    hipDeviceSynchronize();

    print(c, N);

    hipFree(a);
    hipFree(b);
    hipFree(c);
    
    cout<<"caller end\n";
}