#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
#include <array>

#include <thrust/iterator/zip_iterator.h>
#include <thrust/universal_vector.h>

using namespace std;

/*
    Example of thrust zip iterator
*/

int main() 
{
  constexpr int N{5};
  thrust::universal_vector<int> a{N};
  thrust::universal_vector<int> b{N};

  for(int i = 0; i < N; i++)
  {
    a[i] = i;
    b[i] = i * i;
  }

  auto zip = thrust::make_zip_iterator(a.begin(), b.begin());

  auto transform = thrust::make_transform_iterator(zip,
  [] __host__ __device__ (thrust::tuple<int, int> t)
  {
    return abs(thrust::get<0>(t) - thrust::get<1>(t));
  });

  printf("transform = %d\n", *transform);

  transform++;
  transform++;
  
  printf("transform = %d\n", *transform);

  printf("b[%d] = %d\n", 2, b[2]);
}