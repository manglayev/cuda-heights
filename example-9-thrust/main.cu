#include ""
#include "hip/hip_runtime.h"

#include <thrust/execution_policy.h>
#include <thrust/universal_vector.h>
#include <thrust/transform.h>
#include <cstdio>
#include <iostream>
using namespace std;

/*
    Simple C++ transform run on the device and lambda
*/

void playing_with_lambda()
{
    int x = 10;
    auto lambda = [=](){ printf("X = %d;\n", x); };
    printf("x = %d;\n", x);
    lambda();

    auto lambda2 = [=]() { return x * 3; };
    int y = lambda2();
    printf("y = %d;\n", y);
}

int main()
{
    cout << "caller start;\n";    
    
    playing_with_lambda();

    size_t N = 5;
    thrust::universal_vector<int> c {N};
    for(int i = 0; i < c.size(); i++)
    {
        c[i] = i * 2;
        printf("%d ", c[i]);
    }
    printf("\n");

    thrust::transform(thrust::device, c.begin(), c.end(), c.begin(), [] __host__ __device__ (int d) { return d * 3; });

    for(int i = 0; i < c.size(); i++)
    {
        printf("%d ", c[i]);
    }
    printf("\n");
    cout << "caller end;\n";
}