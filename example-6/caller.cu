#include <iostream>

#include "hip/hip_runtime.h"

#define THREADS 64
#define BLOCKS 8

using namespace std;

__global__ void reduction(int *input, int *output)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    __shared__ int sdata[THREADS];
    sdata[threadIdx.x] = input[i];
    __syncthreads();

    for(int s = 1; s < blockDim.x; s*=2)
    {
        if(threadIdx.x % (2 * s) == 0)
            sdata[threadIdx.x] += sdata[threadIdx.x + s];
        __syncthreads(); 
    }
    if(threadIdx.x == 0)
        output[blockIdx.x] = sdata[0];
    __syncthreads();
}
/* SIMPLE CUDA REDUCTION PROGRAM */
int main()
{
    int h_a[THREADS * BLOCKS];
    int h_b[1];
    int *d_a;
    int *d_b;

    for(int i = 0; i < THREADS * BLOCKS; i++)
    {
        h_a[i] = 1;
    }

    hipMalloc((void**)&d_a, THREADS * BLOCKS * sizeof(int));
    hipMalloc((void**)&d_b, BLOCKS * sizeof(int));

    hipMemcpy(d_a, h_a, THREADS * BLOCKS * sizeof(int), hipMemcpyHostToDevice);

    reduction<<<BLOCKS, THREADS>>>(d_a, d_b);
    reduction<<<1, BLOCKS>>>(d_b, d_b);

    hipMemcpy(h_b, d_b, sizeof(int), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    cout << h_b[0]<<"\n";

    hipFree(d_a);
    hipFree(d_b);

}