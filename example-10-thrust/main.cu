
#include <hip/hip_runtime.h>
#include <cstdio>
#include <iostream>
#include <array>

using namespace std;

/*
    Example of count iterator
*/

struct count_iterator
{
    int operator[](int i)
    {
        return i;
    }
};

int main()
{
    cout << "main start;\n";
    count_iterator ci;

    printf("ci[%d] = %d\n", 0, ci[0]);
    printf("ci[%d] = %d\n", 5, ci[5]);
    cout << "main end;\n";
}