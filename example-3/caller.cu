#include "hip/hip_runtime.h"
#include <iostream>
#include ""
#include "hip/hip_runtime.h"

using namespace std;

__global__ void init(int *a, int N)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for(int i = id; i < N; i += stride)
    {
        a[i] = 1;
    }
}

__global__ void sum(int *a, int *b, int *c, int N)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for(int i = id; id < N; i += stride)
    {
        c[i] = a[i] + b[i];
    }
}

void print(int *a, int N)
{
    int c = 0;
    for(int i = 0; i < N; i++)
    {
        cout << a[i] << " ";
        c++;
        if(c % 8 == 0) cout << "\n";
    }
}

int main()
{
    int *a;
    int *b;
    int *c;

    int N = 2<<7;
    size_t size = N * sizeof(int);

    hipMallocManaged(&a, size);
    hipMallocManaged(&b, size);
    hipMallocManaged(&c, size);

    int deviceId;
    int number_of_SMs;

    hipGetDevice(&deviceId);
    hipDeviceGetAttribute(&number_of_SMs, hipDeviceAttributeMultiprocessorCount, deviceId);

    hipMemPrefetchAsync(&a, size, deviceId);
    hipMemPrefetchAsync(&b, size, deviceId);
    hipMemPrefetchAsync(&c, size, deviceId);

    init<<<8, 32>>>(a, N);
    init<<<8, 32>>>(b, N);
    init<<<8, 32>>>(c, N);
      
    sum<<<8, 32>>>(a, b, c, N);

    hipDeviceSynchronize();
    hipMemPrefetchAsync(c, size, hipCpuDeviceId);
    print(c, N);

    //printf("number of multi processors: %d\n", number_of_SMs);
    
    hipFree(a);
    hipFree(b);
    hipFree(c);
}