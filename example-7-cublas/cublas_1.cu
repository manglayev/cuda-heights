#include "hip/hip_runtime.h"
#include ""
#include <hipblas.h>
#include "cublas_1_header.cuh"

//Initialize a vector
void vector_init(float *a)
{
  for(int i = 0; i < N; i++)
  {
    a[i] = (float)(rand() % 100);
  }
}

//Verify the result
void verify_result(float *a, float *b, float *c, float factor)
{
  for(int i = 0; i < N; i++)
  {
    assert(c[i] == factor * a[i] + b[i]);
  }
}
//Verify the result 2
void verify_result(float *d_c, float *h_c)
{
  for(int i = 0; i < N; i++)
  {
    assert(d_c[i] == h_c[i]);
  }
}
float* saxpy(float *a, float *b, float *c, float factor)
{
  for(int i = 0; i < N; i++)
  {
    c[i] = factor * a[i] + b[i];
  }
  return c;
}

//Print N x N matrix
void printArray(float *a)
{
  // loop through array's rows
  for(int i = 0; i < N; i++)
  {
    if(a[i] >= 0)
      std::cout << ' ' << a[i] << ' ';
    else
      std::cout << a[i] << ' ';
  }
  std::cout << '\n'; // start new line of output
}

void wrapper()
{
  printf("STAGE 3 WRAPPER START\n");
  // Vector size
  size_t bytes = N * sizeof(float);
  //Declare vector pointers
  float *h_a, *h_b, *h_c, *h_c_cpu;
  float *d_a, *d_b;

  // Allocate memory
  h_a = (float*)malloc(bytes);
  h_b = (float*)malloc(bytes);
  h_c = (float*)malloc(bytes);
  h_c_cpu = (float*)malloc(bytes);
  
  hipMalloc(&d_a, bytes);
  hipMalloc(&d_b, bytes);

  // Initialize vectors
  vector_init(h_a);
  vector_init(h_b);

  // Create and initialize a new context
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  // Copy the vectors over to the device
  hipblasSetVector(N, sizeof(float), h_a, 1, d_a, 1);
  hipblasSetVector(N, sizeof(float), h_b, 1, d_b, 1);

  // Launch simple saxpy kernel (single precision a * x + y)
  const float scale = 2.0f;
  hipblasSaxpy(handle, N, &scale, d_a, 1, d_b, 1);
  
  //Copy the result vector back
  hipblasGetVector(N, sizeof(float), d_b, 1, h_c, 1);

  //Saxpy CPU version
  h_c_cpu = saxpy(h_a, h_b, h_c_cpu, scale);
  //Print out the result
  printArray(h_c);
  printArray(h_c_cpu);
  
  //Verify the result
  //verify_result(h_a, h_b, h_c, scale, N);
  // Clean up the created handle
  hipblasDestroy(handle);

  // Release allocated memory
  hipFree(d_a);
  hipFree(d_b);

  free(h_a);
  free(h_b);
  free(h_c_cpu);
  printf("STAGE 3 WRAPPER END\n");
}