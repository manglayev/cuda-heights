#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
#include <array>

#include <thrust/iterator/zip_iterator.h>

using namespace std;

/*
    Example of zip and transform iterator
*/

struct zip_i
{
    int *a;
    int *b;

    std::tuple<int, int> operator[](int i)
    {
        return {a[i], b[i]};
    }
};

struct transform_absolute
{
    zip_i z;
    
    int operator[](int i)
    {
        auto [a, b] = z[i];
        return abs(a - b);
    }
};

int main()
{
    cout << "main start;\n";
    std::array<int, 5> a{1, 33, 5, 77, 9};
    std::array<int, 5> b{11, 3, 55, 7, 99};

    zip_i z {a.data(), b.data()};
    transform_absolute t{z};

    printf("absolute difference of a[%d] = %d and b[%d] = %d is %d\n", 2, a[2], 2, b[2], t[2] );

    cout << "main end;\n";
}